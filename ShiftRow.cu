#include "hip/hip_runtime.h"
// Filename: ShiftRow.cpp
// SIUE CS490-005
// Spring 2017
// Authors: David Shimkus and Michael Caponi
//
// NOTE: Byte operations are performed on char arrays.
//
// Revisions:
// 04/19/2017 | DS | Creation.
// 04/24/2017 | DS | Added the shift operators.
// 04/25/2017 | DS | Added the &'s to parameters.  Changed functions to return void.
// 05/01/2017 | MC | Convert to CUDA.

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>
#include <sys/stat.h> //for filesize

using namespace std;

#include "ShiftRow.h"
#include "State.h"

#include "hip/hip_runtime.h"
#include ""

__global__ void ShiftRow(int A[][4], int B[][4])
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		unsigned char tempBytes[4];
		if (j < 4)
		{
			//perform the left shift as dependent upon the row
			tempBytes[j] = input.bytes[i][(j + i) % 4];
		}
		for (int k = 0; k < 4; k++)
		{
			input.bytes[i][k] = tempBytes[k];
		}
	}
}

__global__ void InvShiftRow(int A[][4], int B[][4])
{
	int offset = 4;
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		int tempBytes[4];
		if (j < 4)
		{
			//perform the right shift as dependent upon the row
			tempBytes[j] = A.bytes[i][(j + offset) % 4];
		}
		for (int k = 0; k < 4; k++)
		{
			A.bytes[i][k] = tempBytes[k];
		}
		offset--;
	}

}