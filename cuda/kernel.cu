#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>
#include <sys/stat.h> //for filesize
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>

#include "hip/hip_runtime.h"


using namespace std;

#define BLOCK_SIZE_BIT 128
#define BLOCK_SIZE_CHAR 16

typedef struct State
{
	int bytes[4][4];
}State;

struct block
{
	unsigned char text[BLOCK_SIZE_CHAR]; 
};


__global__ void ByteSub(int A[][4], int B[][4])
{

	const int Matrix_ByteSub[][16] = {
		{ 0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76 },
		{ 0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0 },
		{ 0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15 },
		{ 0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75 },
		{ 0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84 },
		{ 0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf },
		{ 0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8 },
		{ 0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2 },
		{ 0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73 },
		{ 0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb },
		{ 0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79 },
		{ 0xe7, 0xcb, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08 },
		{ 0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a },
		{ 0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e },
		{ 0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe8, 0xce, 0x55, 0x28, 0xdf },
		{ 0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 }
	};

	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		if (j < 4)
		{
			//get the ASCII value as well as its decimal value
			int byteTemp = A[i][j];

			//get the leftmost 4 bits aka the COLUMN
			byteTemp = A[i][j];
			byteTemp = (byteTemp >> 4) & ((1 << 4) - 1); //leftmost 4 bits
			int column = byteTemp;

			//get the rightmost 4 bits aka the ROW
			byteTemp = A[i][j];
			byteTemp = (byteTemp >> 0) & ((1 << 4) - 1); //rightmost 4 bits
			int row = byteTemp;

			//set the original bytes on the passed in matrix to the new bytes
			B[i][j] = Matrix_ByteSub[column][row];
		}
	}
}

__global__ void ShiftRow(int A[][4], int B[][4])
{

	//NEED TO GET A INTO B


	int i = threadIdx.x;
	int j = threadIdx.y;
	int k = threadIdx.y;

	


	if (i < 4)
	{
		unsigned char tempBytes[4];
		if (j < 4)
		{
			//perform the left shift as dependent upon the row
			tempBytes[j] = A[i][(j + i) % 4];
			//B[i][j] = tempBytes[j];
		}
		if (k < 4)
		{
			A[i][k] = tempBytes[k];
			//B[i][k] = A[i][k];
		}
	}

	for(int m = 0; m < 4; m ++)
	{
		for(int n = 0; n < 4; n++)
		{

			B[m][n]=A[m][n];
		}
	}
}


__device__ unsigned char gmul(unsigned char a, unsigned char b)
{
	//Original Author: Sam Trenholme

	//GF(2^8) multiplication/addition
	unsigned char p = 0; // the product of the multiplication
	unsigned char counter;
	unsigned char hi_bit_set;
	for (counter = 0; counter < 8; counter++)
	{
		// if b is odd, then add (XOR) the corresponding a to p (final product = sum of all a's corresponding to odd b's)
		if ((b & 1) == 1)
		{
			p ^= a;
		}

		// GF modulo: if a >= 128, then it will overflow when shifted left, so reduce
		hi_bit_set = (a & 0x80);
		a <<= 1;

		// if a's hi bit had a value of one prior to this rotation, exclusive or a with the hexadecimal number 0x1b
		// 0x1b = x^4 + x^3 + x + 1 of GF(2^8) irreducible polynomial
		if (hi_bit_set == 0x80)
		{
			a ^= 0x1b;
		}

		// rotate b one bit to the right, discarding the low bit, and making the high (eighth from left) bit have a value of zero		
		b >>= 1;
	}
	return p;
}

__global__ void MixColumn(int A[][4], int B[][4])
{
	unsigned char col[4];
	unsigned char copyCol[4];

	int i = threadIdx.x;
	//int j = threadIdx.y;
	//int k = threadIdx.y;

	/*
	if (i < 4)
	{
		if (j < 4)
		{
			col[j] = A[j][i];
		}

		// apply the mixColumn on one column 
		unsigned char copyCol[4];

		if (k < 4)
		{
			copyCol[k] = col[k];
		}

		col[0] = gmul(copyCol[0], 2) ^ gmul(copyCol[1], 3) ^ gmul(copyCol[2], 1) ^ gmul(copyCol[3], 1);
		col[1] = gmul(copyCol[0], 1) ^ gmul(copyCol[1], 2) ^ gmul(copyCol[2], 3) ^ gmul(copyCol[3], 1);
		col[2] = gmul(copyCol[0], 1) ^ gmul(copyCol[1], 1) ^ gmul(copyCol[2], 2) ^ gmul(copyCol[3], 3);
		col[3] = gmul(copyCol[0], 3) ^ gmul(copyCol[1], 1) ^ gmul(copyCol[2], 1) ^ gmul(copyCol[3], 2);

		// put the values back into the state 
		if (j < 4)
		{
			B[j][i] = col[j];
		}
	}
	*/

	if (i < 4)
	{
		for(int m = 0; m < 4; m++)
		{
			copyCol[m] = A[m][i];
		}

		// apply the mixColumn on one column 

		col[0] = gmul(copyCol[0], 2) ^ gmul(copyCol[1], 3) ^ gmul(copyCol[2], 1) ^ gmul(copyCol[3], 1);
		col[1] = gmul(copyCol[0], 1) ^ gmul(copyCol[1], 2) ^ gmul(copyCol[2], 3) ^ gmul(copyCol[3], 1);
		col[2] = gmul(copyCol[0], 1) ^ gmul(copyCol[1], 1) ^ gmul(copyCol[2], 2) ^ gmul(copyCol[3], 3);
		col[3] = gmul(copyCol[0], 3) ^ gmul(copyCol[1], 1) ^ gmul(copyCol[2], 1) ^ gmul(copyCol[3], 2);

		// put the values back into the state 
		for(int n = 0; n < 4; n++)
		{
			B[n][i] = col[n];
		}
	}
}


int main()
{
	int i, j;
	dim3 threadsPerBlock(4, 4);

	 int test_matrix[4][4] = {
		{ 0x00, 0x12, 0x0c, 0x08 },
		{ 0x04, 0x04, 0x00, 0x23 },
		{ 0x12, 0x12, 0x13, 0x19 },
		{ 0x14, 0x00, 0x11, 0x19 }
	 };

	State test;
	for (int i = 0; i < 4; i++) {
		for (int j  =0; j < 4; j++) {
			test.bytes[i][j] = test_matrix[i][j];
		}
	}

	int BS[4][4];
	int SR[4][4];
	int MC[4][4];

	int(*d_test)[4];
	int(*d_BS)[4];
	int(*d_SR)[4];
	int(*d_MC)[4];

	hipMalloc((void**)&d_test, (4 * 4) * sizeof(int));
	hipMalloc((void**)&d_BS, (4 * 4) * sizeof(int));
	hipMalloc((void**)&d_SR, (4 * 4) * sizeof(int));
	hipMalloc((void**)&d_MC, (4 * 4) * sizeof(int));

	hipMemcpy(d_test, &test, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_BS, BS, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_SR, SR, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_MC, MC, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);

	/*------------------BYTESUB--------------------*/
	ByteSub <<<4, threadsPerBlock >>>(d_test, d_BS);

	hipMemcpy(&test, d_test, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(BS, d_BS, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);

	/*------------------SHIFTROW--------------------*/
	State bytesub;
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			bytesub.bytes[i][j] = BS[i][j];
		}
	}

	int(*d_bytesub)[4];
	hipMalloc((void**)&d_bytesub, (4 * 4) * sizeof(int));

	hipMemcpy(d_bytesub, &bytesub, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_SR, SR, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);

	ShiftRow<<<4, threadsPerBlock >>>(d_bytesub, d_SR);

	hipMemcpy(&bytesub, d_bytesub, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(SR, d_SR, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);

	/*------------------MIXCOLUMN--------------------*/
	State shiftrow;
	for (int i = 0; i < 4; i++) {
		for (int j = 0; j < 4; j++) {
			shiftrow.bytes[i][j] = SR[i][j];
		}
	}

	int(*d_shiftrow)[4];
	hipMalloc((void**)&d_shiftrow, (4 * 4) * sizeof(int));

	hipMemcpy(d_shiftrow, &shiftrow, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_SR, SR, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);

	MixColumn <<<4, threadsPerBlock >>>(d_shiftrow, d_MC);

	hipMemcpy(&shiftrow, d_shiftrow, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(MC, d_MC, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);


	printf("test = \n");
	for (i = 0; i<4; i++) {
		for (j = 0; j<4; j++) {
			//printf("%d ", test.bytes[i][j]);
			cout << hex << test.bytes[i][j] << endl;
		}
		printf("\n\n");
	}

	printf("BS = \n");
	for (i = 0; i<4; i++) {
		for (j = 0; j<4; j++) {
			//printf("%d ", BS[i][j]);
			cout << hex << BS[i][j] << endl;
		}
		printf("\n\n");
	}

	printf("SR = \n");
	for (i = 0; i<4; i++) {
		for (j = 0; j<4; j++) {
			//printf("%d ", SR[i][j]);
			cout << hex << SR[i][j] << endl;
		}
		printf("\n\n");
	}

	printf("MC = \n");
	for (i = 0; i<4; i++) {
		for (j = 0; j<4; j++) {
			//printf("%d ", MC[i][j]);
			cout << hex << MC[i][j] << endl;
		}
		printf("\n\n");
	}

	hipFree(d_test);
	hipFree(d_BS);
	hipFree(d_SR);
	hipFree(d_MC);

	return 0;
}
