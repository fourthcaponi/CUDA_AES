#include "hip/hip_runtime.h"
// Filename: KeyExpansion.cpp
// SIUE CS490-005
// Spring 2017
// Authors: David Shimkus and Michael Caponi
//
// NOTE: Byte operations are performed on char arrays.
//
// Revisions:
// 04/26/2017 | DS | Creation.
// 05/01/2017 | MC | cuda implementation
// 05/01/2017 | MC | Convert to CUDA.

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>

#include "State.h"
#include "Matrices.h"

#include "hip/hip_runtime.h"
#include ""

using namespace std;

__device__ void RotWord(Word &word)
{
	unsigned char tempBytes[4];
	for (int i = 0; i < 4; i++)
	{
		//place the appropriate left shifted bytes into the temp variable
		tempBytes[i] = word.bytes[(i + 1) % 4];
	}
	for (int k = 0; k < 4; k++)
	{
		//place the temp var's data into the original input word
		word.bytes[k] = tempBytes[k];
	}
}

__device__ void SubWord(Word &word)
{
	for (int i = 0; i < 4; i++)
	{
		//get the ASCII value as well as its decimal value
		int byteTemp = word.bytes[i];

		//get the leftmost 4 bits aka the COLUMN
		byteTemp = (byteTemp >> 4) & ((1 << 4) - 1); //leftmost 4 bits
		int column = byteTemp;

		//get the rightmost 4 bits aka the ROW
		byteTemp = word.bytes[i];
		byteTemp = (byteTemp >> 0) & ((1 << 4) - 1); //rightmost 4 bits
		int row = byteTemp;

		//set the original bytes on the passed in matrix to the new bytes
		word.bytes[i] = Matrix_Rijndael[column][row];
	}
}

__global__ void ke_128(Word *keyWords, int numKeyWords) 
{
	Word tempWord;
	for (int i = 4; i < numKeyWords; i++)
	{
		if (i % 4 != 0)
		{
			for (int j = 0; j < 4; j++)
			{
				keyWords[i].bytes[j] = keyWords[i - 1].bytes[j] ^ keyWords[i - 4].bytes[j];
			}
		}
		else
		{
			//function g
			for (int k = 0; k < 4; k++)
			{
				tempWord.bytes[k] = keyWords[i - 1].bytes[k]; //initially populate the tempword		
			}
			//perform the rotation
			RotWord(tempWord);
			//apply the subword
			SubWord(tempWord);

			//XOR the [0]'th byte with RCon[i/4]
			tempWord.bytes[0] ^= Matrix_RCon[i / 4];

			//apply the addition (XOR?)of 't' aka tempWord
			for (int l = 0; l < 4; l++)
			{
				keyWords[i].bytes[l] = tempWord.bytes[l] ^ keyWords[i - 4].bytes[l];
			}
		}
	}
}

__global__ void ke_192(Word *keyWords, int numKeyWords)
{
	//note: this is basically the same as 128 bit except for 6 word "groups" instead of 4
	Word tempWord;
	for (int i = 4; i < numKeyWords; i++)
	{
		if (i % 6 != 0)
		{
			for (int j = 0; j < 4; j++)
			{
				//note the keyWords[i-6] here instead of [i-4]
				keyWords[i].bytes[j] = keyWords[i - 1].bytes[j] ^ keyWords[i - 6].bytes[j];
			}
		}
		else
		{
			//function g
			for (int k = 0; k < 4; k++)
			{
				//note 'tempword' is the 't' variable from the book
				tempWord.bytes[k] = keyWords[i - 1].bytes[k]; //initially populate the tempword		
			}
			//perform the rotation
			RotWord(tempWord);
			//apply the subword
			SubWord(tempWord);

			//XOR the [0]'th byte it with Rconi/4
			tempWord.bytes[0] ^= Matrix_RCon[i / 4];

			//apply the addition (XOR?)of 't' aka tempWord
			for (int l = 0; l < 4; l++)
			{
				keyWords[i].bytes[l] = tempWord.bytes[l] ^ keyWords[i - 6].bytes[l];
			}
		}
	}
}

__global__ void ke_256(Word *keyWords, int numKeyWords) 
{

	

	Word tempWord;
	if (int i = 8; i < numKeyWords; i++)
	{
		if (i % 8 != 0)
		{
			//check for the additional step needed for 256
			//this is subsection 2.c. of pg 212 from the textbook
			if (i % 4 == 0)
			{
				for (int j = 0; j < 4; j++)
				{
					tempWord.bytes[j] = keyWords[i - 1].bytes[j];
				}
				//perform the SubWord
				SubWord(tempWord);

				for (int j = 0; j < 4; j++)
				{
					tempWord.bytes[j] ^= keyWords[i - 8].bytes[j];
				}
			}
			else
			{
				for (int j = 0; j < 4; j++)
				{
					keyWords[i].bytes[j] = keyWords[i - 1].bytes[j] ^ keyWords[i - 8].bytes[j];

				}
			}
		}
		else
		{
			for (int k = 0; k < 4; k++)
			{
				tempWord.bytes[k] = keyWords[i - 1].bytes[k]; //initially populate the tempword		
			}
			//perform the rotation
			RotWord(tempWord);
			//apply the subword
			SubWord(tempWord);
			//XOR it with Rcon[i/4]
			for (int k = 0; k < 4; k++)
			{
				tempWord.bytes[k] ^= Matrix_RCon[i / 4];
			}
			//apply the addition (XOR?)of 't' aka tempWord
			for (int l = 0; l < 4; l++)
			{
				keyWords[i].bytes[l] = tempWord.bytes[l] ^ keyWords[i - 8].bytes[l];
			}
		}
	}
}