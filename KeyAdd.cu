#include "hip/hip_runtime.h"
// Filename: KeyAdd.cpp
// SIUE CS490-005
// Spring 2017
// Authors: David Shimkus and Michael Caponi
//
// NOTE: Byte operations are performed on char arrays.
//
// Revisions:
// 04/19/2017 | DS | Creation.
// 04/26/2017 | DS | Work on KeyAdd
// 04/27/2017 | DS | Updated to reflect Cipher() 
// 04/28/2017 | DS | Modified to reflect latest Cipher()
// 05/01/2017 | MC | Convert to CUDA.

using namespace std;

#include "State.h"

#include "hip/hip_runtime.h"
#include ""

//note KeyAdd is the inverse of itself
__device__ void KeyAdd(State &input, Word(&keyWords)[60], size_t size, int round)
{
	int counter = round * 4; //appropriate offset

	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		if (j < 4)
		{

			//perform the XOR operation
			//note the order of [j] then [i]
			//input.bytes[j][i] ^= keyWords[counter].bytes[j];
			//temp = keyWords[counter].bytes[j];
			//input.bytes[j][i] ^= keyWords[counter].bytes[j];

			input.bytes[j][i] ^= keyWords[counter].bytes[j];
			
		}
		counter++;
	}
}