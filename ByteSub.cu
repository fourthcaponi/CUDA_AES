// Filename: ByteSub.cpp
// SIUE CS490-005
// Spring 2017
// Authors: David Shimkus and Michael Caponi
//
// NOTE: Byte operations are performed on char arrays.
//
// Revisions:
// 04/19/2017 | DS | Creation.
// 04/24/2017 | DS | Added the hex/dec stuff, rips each byte into 4 bit chunks.
// 04/24/2017 | DS | Work on InvByteSub.  
// 04/26/2017 | DS | Code cleanup.
// 05/01/2017 | MC | Convert to CUDA.

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>

using namespace std;

#include "ByteSub.h"
#include "State.h"
#include "Matrices.h"
#include "KeyExpansion.h"

#include "hip/hip_runtime.h"
#include ""

__device__ void ByteSub(State &input)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		if (j < 4)
		{
			//get the ASCII value as well as its decimal value
			int byteTemp = input.bytes[i][j];

			//get the leftmost 4 bits aka the COLUMN
			byteTemp = input.bytes[i][j];
			byteTemp = (byteTemp >> 4) & ((1 << 4) - 1); //leftmost 4 bits
			int column = byteTemp;

			//get the rightmost 4 bits aka the ROW
			byteTemp = input.bytes[i][j];
			byteTemp = (byteTemp >> 0) & ((1 << 4) - 1); //rightmost 4 bits
			int row = byteTemp;

			//set the original bytes on the passed in matrix to the new bytes
			input.bytes[i][j] = Matrix_ByteSub[column][row];
		}
	}
}

__device__ void InvByteSub(State &input)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		if (j < 4)
		{
			//get the ASCII value as well as its decimal value
			int byteTemp = input.bytes[i][j];

			//get the leftmost 4 bits aka the COLUMN
			byteTemp = input.bytes[i][j];
			byteTemp = (byteTemp >> 4) & ((1 << 4) - 1); //leftmost 4 bits
			int column = byteTemp;

			//get the rightmost 4 bits aka the ROW
			byteTemp = input.bytes[i][j];
			byteTemp = (byteTemp >> 0) & ((1 << 4) - 1); //rightmost 4 bits
			int row = byteTemp;

			//set the original bytes on the passed in matrix to the new bytes
			input.bytes[i][j] = Matrix_InvByteSub[column][row];
		}
	}
}

