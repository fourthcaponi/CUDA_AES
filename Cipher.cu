#include "hip/hip_runtime.h"
// Filename: Cipher.cpp
// SIUE CS490-005
// Spring 2017
// Authors: David Shimkus and Michael Caponi
//
// This is where the magic happens.
//
// Revisions:
// 04/27/2017 | DS | Creation. Moved from Main.cpp for program flow.
// 04/28/2017 | DS | Functions now accept states instead of blocks for input.
// 04/29/2017 | DS | Changed the way the keyWords was getting passed in.
// 05/01/2017 | MC | Convert to CUDA.

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>

using namespace std;

#include "ByteSub.h"
#include "KeyAdd.h"
#include "MixColumn.h"
#include "ShiftRow.h"
#include "State.h"
#include "Matrices.h"

#include "hip/hip_runtime.h"
#include ""

//depending on the passed in round index will determine 
//which indeces of keyWords we will use

//note this "roundIndex WILL go all the way up to numRounds"
__global__ void Cipher(State &input, Word(&keyWords)[60], size_t size, int roundIndex, int numRounds)
{
	if (roundIndex == 0)
	{
		//pre-round transformation
		KeyAdd(input, keyWords, size, 0);
	}
	else if (roundIndex == numRounds)
	{
		ByteSub(input);
		ShiftRow(input);
		KeyAdd(input, keyWords, size, roundIndex);
	}
	else
	{
		ByteSub(input);
		ShiftRow(input);
		MixColumn(input);
		KeyAdd(input, keyWords, size, roundIndex);
	}

	cout << "---- Cipher Output Round #" << dec << roundIndex << " ----\n";
	input.print();

	cout << "---- Key for Round #" << dec << roundIndex << " ----\n";
	for(int i = roundIndex * 4; i < roundIndex*4+4; i ++)
	{
		keyWords[i].print();
	}

}

//note the roundIndex is BACKWARDS from how Dr. Gamage described it in class (?)
__global__ void Decrypt(State &input, Word(&keyWords)[60], size_t size, int roundIndex, int numRounds)
{
	if (roundIndex == 0)
	{
		KeyAdd(input, keyWords, size, numRounds);
		InvShiftRow(input);
		InvByteSub(input);
	}
	else if (roundIndex == numRounds)
	{
		//pre-round transformation
		KeyAdd(input, keyWords, size, 0);
	}
	else
	{
		KeyAdd(input, keyWords, size, numRounds - roundIndex);
		InvMixColumn(input);
		InvShiftRow(input);
		InvByteSub(input);
	}
}

