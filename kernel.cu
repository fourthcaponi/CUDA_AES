#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>
#include <sys/stat.h> //for filesize
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define BLOCK_SIZE_BIT 128
#define BLOCK_SIZE_CHAR 16
#define N 512;
#define GRID_SIZE 1;



struct State
{
	unsigned char bytes[4][4];
	int ByteSub_values[4][4];

	void print()
	{
		cout << endl;
		cout << "{ { " << setfill('0') << setw(2) << hex << (int)bytes[0][0] << ", " << setfill('0') << setw(2) << hex << (int)bytes[0][1] << ", " << setfill('0') << setw(2) << hex << (int)bytes[0][2] << ", " << setfill('0') << setw(2) << hex << (int)bytes[0][3] << " },\n";
		cout << "  { " << setfill('0') << setw(2) << hex << (int)bytes[1][0] << ", " << setfill('0') << setw(2) << hex << (int)bytes[1][1] << ", " << setfill('0') << setw(2) << hex << (int)bytes[1][2] << ", " << setfill('0') << setw(2) << hex << (int)bytes[1][3] << " },\n";
		cout << "  { " << setfill('0') << setw(2) << hex << (int)bytes[2][0] << ", " << setfill('0') << setw(2) << hex << (int)bytes[2][1] << ", " << setfill('0') << setw(2) << hex << (int)bytes[2][2] << ", " << setfill('0') << setw(2) << hex << (int)bytes[2][3] << " },\n";
		cout << "  { " << setfill('0') << setw(2) << hex << (int)bytes[3][0] << ", " << setfill('0') << setw(2) << hex << (int)bytes[3][1] << ", " << setfill('0') << setw(2) << hex << (int)bytes[3][2] << ", " << setfill('0') << setw(2) << hex << (int)bytes[3][3] << " } }\n\n";
	}

	void printAscii()
	{
		cout << endl;
		cout << "{ { " << (char)bytes[0][0] << ", " << (char)bytes[0][1] << ", " << (char)bytes[0][2] << ", " << (char)bytes[0][3] << " },\n";
		cout << "  { " << (char)bytes[1][0] << ", " << (char)bytes[1][1] << ", " << (char)bytes[1][2] << ", " << (char)bytes[1][3] << " },\n";
		cout << "  { " << (char)bytes[2][0] << ", " << (char)bytes[2][1] << ", " << (char)bytes[2][2] << ", " << (char)bytes[2][3] << " },\n";
		cout << "  { " << (char)bytes[3][0] << ", " << (char)bytes[3][1] << ", " << (char)bytes[3][2] << ", " << (char)bytes[3][3] << " } }\n\n";

	}
};

struct block
{
	unsigned char text[BLOCK_SIZE_CHAR]; //divide by 8 b/c 1 char = 1 byte
};


__global__ void ByteSub(int A[][4], int B[][4])
{

	const int Matrix_ByteSub[][16] = {
		{ 0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76 },
		{ 0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0 },
		{ 0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15 },
		{ 0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75 },
		{ 0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84 },
		{ 0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf },
		{ 0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8 },
		{ 0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2 },
		{ 0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73 },
		{ 0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb },
		{ 0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79 },
		{ 0xe7, 0xcb, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08 },
		{ 0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a },
		{ 0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e },
		{ 0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe8, 0xce, 0x55, 0x28, 0xdf },
		{ 0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 }
	};

	int i = threadIdx.x;
	int j = threadIdx.y;

	if (i < 4)
	{
		if (j < 4)
		{
			//get the ASCII value as well as its decimal value
			int byteTemp = A[i][j];

			//get the leftmost 4 bits aka the COLUMN
			byteTemp = A[i][j];
			byteTemp = (byteTemp >> 4) & ((1 << 4) - 1); //leftmost 4 bits
			int column = byteTemp;

			//get the rightmost 4 bits aka the ROW
			byteTemp = A[i][j];
			byteTemp = (byteTemp >> 0) & ((1 << 4) - 1); //rightmost 4 bits
			int row = byteTemp;

			//set the original bytes on the passed in matrix to the new bytes
			B[i][j] = Matrix_ByteSub[column][row];
		}
	}
}

int main()
{

	State A = {
		{ 0x00, 0x12, 0x0c, 0x08 },
		{ 0x04, 0x04, 0x00, 0x23 },
		{ 0x12, 0x12, 0x13, 0x19 },
		{ 0x14, 0x00, 0x11, 0x19 }
	};

	int B[4][4];

	int(*pA)[4];
	int(*pB)[4];
	hipMalloc((void**)&pA, (4*4) * sizeof(int));
	hipMalloc((void**)&pB, (4*4) * sizeof(int));


	hipMemcpy(pA, A, (4*4) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pB, B, (4 * 4) * sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(4, 4);
	ByteSub<<<4, threadsPerBlock>>>(pA, pB);

	hipMemcpy(A, pA, (4*4) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(B, pB, (4 * 4) * sizeof(int), hipMemcpyDeviceToHost);


	int i, j; printf("B = \n");
	for (i = 0; i<4; i++) {
		for (j = 0; j<4; j++) {
			printf("%d ", B[i][j]);
		}
		printf("\n");
	}

	hipFree(pA);
	hipFree(pB);

    return 0;
}
